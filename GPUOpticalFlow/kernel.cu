#include "hip/hip_runtime.h"
#include "atlimage.h"
#include "hip/hip_runtime.h"
#include ""
#include "limits.h"
#include "time.h"
#include "windows.h"
#include <stdio.h>
#include <stdlib.h>

#define WIDTH 640
#define HEIGHT 480
#define DEBUG false
#define RENDER_GAIN 20
#define ZERO_PRIOR 1000

 void renderOpticalFlow(int flow[WIDTH * HEIGHT * 2], CImage *image)
{
	for (int i = 0; i < WIDTH; i++)
		for (int j = 0; j < HEIGHT; j++)
		{
			byte r = 0;
			byte g = (byte) abs(RENDER_GAIN * flow[(i * HEIGHT * 2) + (j * 2) + 1]);
			byte b = (byte) abs(RENDER_GAIN * flow[(i * HEIGHT * 2) + (j * 2) + 0]);
			COLORREF color = RGB(r,g,b);
			(*image).SetPixel(i, j, color);
		}
}

 __device__ void threshold(int *value, int max)
 {
	 if (*value < 0)
		 *value = 0;

	 if (*value > max)
		 *value = max;
 }

 //Calculate match value for a fixed x,y location and shift value
__device__ void sumSquareErrorOffsetKernel(int frame1[WIDTH * HEIGHT], int frame2[WIDTH * HEIGHT], int x, int y, int dx, int dy, int *sse, int block_width)
 {
	 int x_start = x - block_width / 2;
	 int y_start = y - block_width / 2;
	 int x_end = x_start + block_width;
	 int y_end = y_start + block_width;

	 threshold(&x, WIDTH);
	 threshold(&y, HEIGHT);

	 int sumSquareError = 0;
	 for (int i = x_start; i < x_end; i++)
		 for (int j = y_start; j < y_end; j++)
		 {
			 bool shifted_point_in_bounds = i + dx >= 0 && i + dx < WIDTH && j + dy >= 0 && j + dy < HEIGHT;
			 bool original_point_in_bounds = i >= 0 && i < WIDTH && j >= 0 && j < HEIGHT;
			 if (shifted_point_in_bounds && original_point_in_bounds)
				 sumSquareError += (frame1[i * HEIGHT + j] - frame2[(i + dx) * HEIGHT + j + dy])*(frame1[i * HEIGHT + j] - frame2[(i + dx) * HEIGHT + j + dy]);
			 else 
				 sumSquareError += 3 * 255;
		 }
			 
		
	 *sse = sumSquareError;
 }

//Find maximum match value at a specified pixel for all possible shifts 
__global__ void opticalFlowPixelKernel(int frame1[WIDTH * HEIGHT], int frame2[WIDTH * HEIGHT], int flow[WIDTH * HEIGHT * 2], int *max_shift, int *block_width)
{
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int best_dx=0;
	int best_dy=0;
	int best_match = INT_MAX;
	if (x >= WIDTH || y >= HEIGHT) return;

	for (int dx = -(*max_shift); dx < (*max_shift); dx++)
		for (int dy = -(*max_shift); dy < (*max_shift); dy++)
		{
			int sse = INT_MAX;

			sumSquareErrorOffsetKernel(frame1, frame2, x, y, dx, dy, &sse, (*block_width));

			if (dx == 0)
				sse -= ZERO_PRIOR;

			if (dy == 0)
				sse -= ZERO_PRIOR;

			if (sse < best_match)
			{
				best_match = sse;
				best_dx = dx;
				best_dy = dy;
			}
		}

	flow[(x * HEIGHT * 2) + (y * 2) + 0] = best_dx;
	flow[(x * HEIGHT * 2) + (y * 2) + 1] = best_dy;
}

 hipError_t opticalFlow(const int *frame1,const int *frame2, int *flow, int max_shift, int block_width)
 {
	 int *frame1_in = 0;
	 int *frame2_in = 0;
	 int *flow_out = 0;
	 int *max_shift_in = 0;
	 int *block_width_in = 0;

	 hipError_t cudaStatus;
	 int size = WIDTH * HEIGHT * sizeof(int);

	 dim3 dimBlock(32, 32);
	 dim3 dimGrid;
	 dimGrid.x = (WIDTH + dimBlock.x - 1) / dimBlock.x;  /*< Greater than or equal to image width */
	 dimGrid.y = (HEIGHT + dimBlock.y - 1) / dimBlock.y; /*< Greater than or equal to image height */

	 // Choose which GPU to run on, change this on a multi-GPU system.
	 cudaStatus = hipSetDevice(0);
	 if (cudaStatus != hipSuccess) {
		 fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		 goto Error;
	 }

	 // Allocate GPU buffers for three arrays (two input, one output plus single element parameters)
	 cudaStatus = hipMalloc((void**)&frame1_in, size);
	 if (cudaStatus != hipSuccess) {
		 fprintf(stderr, "hipMalloc failed!");
		 goto Error;
	 }

	 cudaStatus = hipMalloc((void**)&frame2_in, size);
	 if (cudaStatus != hipSuccess) {
		 fprintf(stderr, "hipMalloc failed!");
		 goto Error;
	 }

	 cudaStatus = hipMalloc((void**)&flow_out, 8 * size);
	 if (cudaStatus != hipSuccess) {
		 fprintf(stderr, "hipMalloc failed!");
		 goto Error;
	 }

	 cudaStatus = hipMalloc((void**)&max_shift_in, sizeof(int));
	 if (cudaStatus != hipSuccess) {
		 fprintf(stderr, "hipMalloc failed!");
		 goto Error;
	 }

	 cudaStatus = hipMalloc((void**)&block_width_in, sizeof(int));
	 if (cudaStatus != hipSuccess) {
		 fprintf(stderr, "hipMalloc failed!");
		 goto Error;
	 }

	 // Copy input vectors from host memory to GPU buffers.
	 cudaStatus = hipMemcpy(frame1_in, frame1, size, hipMemcpyHostToDevice);
	 if (cudaStatus != hipSuccess) {
		 fprintf(stderr, "hipMemcpy failed!");
		 goto Error;
	 }

	 cudaStatus = hipMemcpy(frame2_in, frame2, size, hipMemcpyHostToDevice);
	 if (cudaStatus != hipSuccess) {
		 fprintf(stderr, "hipMemcpy failed!");
		 goto Error;
	 }

	 cudaStatus = hipMemcpy(max_shift_in, &max_shift, sizeof(int), hipMemcpyHostToDevice);
	 if (cudaStatus != hipSuccess) {
		 fprintf(stderr, "hipMemcpy failed!");
		 goto Error;
	 }

	 cudaStatus = hipMemcpy(block_width_in, &block_width, sizeof(int), hipMemcpyHostToDevice);
	 if (cudaStatus != hipSuccess) {
		 fprintf(stderr, "hipMemcpy failed!");
		 goto Error;
	 }

	 // Launch a kernel on the GPU with one thread for each element.
	 opticalFlowPixelKernel<<< dimGrid, dimBlock >>>(frame1_in, frame2_in, flow_out, max_shift_in, block_width_in);

	 // Check for any errors launching the kernel
	 cudaStatus = hipGetLastError();
	 if (cudaStatus != hipSuccess) {
		 fprintf(stderr, "opticalFlowPixelKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		 goto Error;
	 }

	 // hipDeviceSynchronize waits for the kernel to finish, and returns
	 // any errors encountered during the launch.
	 cudaStatus = hipDeviceSynchronize();
	 if (cudaStatus != hipSuccess) {
		 fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching opticalFlowPixelKernel!\n", cudaStatus);
		 fprintf(stderr, "%s\n", hipGetErrorString(cudaStatus));
		 goto Error;
	 }

	 // Copy output vector from GPU buffer to host memory.
	 cudaStatus = hipMemcpy(flow, flow_out, 2 * size, hipMemcpyDeviceToHost);
	 if (cudaStatus != hipSuccess) {
		 fprintf(stderr, "hipMemcpy failed!");
		 goto Error;
	 }

 Error:
	 hipFree(frame1_in);
	 hipFree(frame2_in);
	 hipFree(flow_out);

	 return cudaStatus;
 }

void CImageToArray(CImage image, int imarray[WIDTH * HEIGHT])
{
	for (int i = 0; i < WIDTH; i++)
		for (int j = 0; j < HEIGHT; j++)
		{
			COLORREF color = image.GetPixel(i, j);
			imarray[i * HEIGHT + j] = GetBValue(color) + GetRValue(color) + GetGValue(color);
		}
}


int frame1_array[WIDTH * HEIGHT];
int frame2_array[WIDTH * HEIGHT];
int flow_array[WIDTH * HEIGHT * 2];

//Command line parameters: MAX_SHIFT BLOCK_WIDTH
//ex:
//GPUOpticalFlow.exe 20 10
int main(int argc, char *argv[])
{
	CImage frame1;
	CImage frame2;
	CImage flow = CImage();
	
	char* frame1_path = argv[3];
	char* frame2_path = argv[4];
	//Code to load/create image goes here
	frame1.Load(_T(frame1_path));
	frame2.Load(_T(frame2_path));

	flow.Create(WIDTH, HEIGHT, 24); //May want to check return value (0 -> success)

	CImageToArray(frame1, frame1_array);
	CImageToArray(frame2, frame2_array);

	if (argc != 5)
	{
		printf("Wrong argument count - expected 4 (MAX_SHIFT BLOCK_WIDTH Frame1Path Frame2Path)");
		return 1; //Code 1: Wrong parameters
	}
	int max_shift = atoi(argv[1]);
	int block_width = atoi(argv[2]);

	hipError_t cudaStatus = opticalFlow(frame1_array, frame2_array, flow_array, max_shift, block_width);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	renderOpticalFlow(flow_array, &flow);

	flow.Save(_T("opticalflow.png"));

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

